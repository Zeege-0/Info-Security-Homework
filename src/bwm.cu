#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include <iostream>
#include "myutils.cpp"
#include "mmd.cu"
#include "mydct.cu"
#include "constants.h"

struct myhandle {
    hipsolverHandle_t solver;
    hipblasHandle_t blas;
    int lwork;
    float *work;
};


void gesvd(size_t rows, size_t cols, float *A, size_t lda, float *S, float *U, size_t ldu, float *V, size_t ldv, hipStream_t stream=0){

    hipsolverHandle_t cusolverHandle;
    hipsolverGesvdjInfo_t gesvdinfo;
    int lwork;
    // float *work;
    int batch_size = (rows / TILE_DIM) * (cols / TILE_DIM);

    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverHandle));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverHandle, stream));
    CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdinfo));

    hipsolverDnSgesvdjBatched_bufferSize(cusolverHandle, 
                                 HIPSOLVER_EIG_MODE_VECTOR,
                                 rows, cols, 
                                 A, lda, S, U, ldu, V, ldv,
                                 &lwork, gesvdinfo, batch_size);

}


void tiled_add_wm(size_t rows, size_t cols, float *A, size_t lda, float *res, size_t ldres, float *workspace, hipStream_t stream=0){

    dct_a100_best_param(rows, cols, A, lda, workspace, ldres, stream);



    idct_a100_best_param(rows, cols, workspace, lda, res, ldres, stream);

}


__global__ void gpu_trans_and_pack_continguous(size_t rows, size_t cols, float *A, size_t lda, float *C, size_t ldc){

    // shared memory size equals to blockDim
    extern __shared__ float sA[];

    int tile_id = threadIdx.x + blockIdx.x * blockDim.x;
    int tile_per_row = cols / TILE_DIM;
    int num_tiles = (rows / TILE_DIM) * (cols / TILE_DIM);
    
    // grid stride loop
#pragma unroll
    for(; tile_id < num_tiles; tile_id += gridDim.x){

        // compute the starting address of current tile in A
        int tile_x = tile_id / tile_per_row;
        int tile_y = tile_id % tile_per_row;
        const float *tile_ptr_to_A = &A[tile_x * TILE_DIM * lda + tile_y * TILE_DIM];
        float *tile_ptr_to_shared = &sA[threadIdx.x * TILE_DIM * TILE_DIM];
        float *tile_ptr_to_res = &C[tile_id * TILE_DIM * TILE_DIM];
        
        // copy to shared memory
        tile_ptr_to_shared[threadIdx.y + threadIdx.z * TILE_DIM] = 
                 tile_ptr_to_A[IDX(threadIdx.y, threadIdx.z, lda)]; // note that leading dimension is cols
        __syncthreads();

        tile_ptr_to_res[threadIdx.y * TILE_DIM + threadIdx.z] = tile_ptr_to_shared[threadIdx.y * TILE_DIM + threadIdx.z];
    }
}

__global__ void gpu_unpack_and_trans(size_t rows, size_t cols, const float *A, size_t lda, float *C, size_t ldc){
    // shared memory size equals to blockDim
    extern __shared__ float sA[];

    int tile_id = threadIdx.x + blockIdx.x * blockDim.x;
    int tile_per_row = cols / TILE_DIM;
    int num_tiles = (rows / TILE_DIM) * (cols / TILE_DIM);
    
    // grid stride loop
#pragma unroll
    for(; tile_id < num_tiles; tile_id += gridDim.x){

        // compute the starting address of current tile in A
        int tile_x = tile_id / tile_per_row;
        int tile_y = tile_id % tile_per_row;
        float *tile_ptr_to_A = &C[tile_x * TILE_DIM * lda + tile_y * TILE_DIM];
        float *tile_ptr_to_shared = &sA[threadIdx.x * TILE_DIM * TILE_DIM];
        const float *tile_ptr_to_res = &A[tile_id * TILE_DIM * TILE_DIM];
        
        tile_ptr_to_shared[threadIdx.y * TILE_DIM + threadIdx.z] = tile_ptr_to_res[threadIdx.y * TILE_DIM + threadIdx.z];
        // copy to shared memory
        tile_ptr_to_A[IDX(threadIdx.y, threadIdx.z, lda)] = tile_ptr_to_shared[threadIdx.y + threadIdx.z * TILE_DIM];
        __syncthreads();

        // printf("(%d, %d, %d): %d\n", tile_id, threadIdx.y, threadIdx.z, tile_x * TILE_DIM * lda + tile_y * TILE_DIM);

    }
}


int mtxtp_a100_best_param(bool input, size_t rows, size_t cols, float *A, size_t lda, float *C, size_t ldc, hipStream_t stream=0){
    dim3 dimGrid(1024);
    dim3 dimgBlock(8, TILE_DIM, TILE_DIM);
    size_t smemSize = TILE_DIM * TILE_DIM * sizeof(int);
    __TIMER_START__(dur);
    if (input) {
        gpu_trans_and_pack_continguous<<<dimGrid, dimgBlock, smemSize, stream>>>(rows, cols, A, lda, C, ldc);
    } else {
        gpu_unpack_and_trans<<<dimGrid, dimgBlock, smemSize, stream>>>(rows, cols, A, lda, C, ldc);
    }
    __TIMER_STOP__(dur);
    std::cout << "Trans: " << dur << std::endl;
}




int main(int argc, char **argv){

    int device = 0;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    std::cout << "Using device " << device << " " << prop.name << std::endl;

    // int N = atoi(argv[1]);
    int rows = atoi(argv[1]);
    int cols = atoi(argv[2]);

    float *A, *AT, *U, *S, *V;
    float *pyU, *pyV, *inv;
    int *info;
    int lda = rows;
    // int ldu = rows;
    // int ldv = rows;
    int lda_T = cols;
    // int ldu_T = cols;
    // int ldv_T = cols;

    hipStream_t stream = NULL;
    hipblasHandle_t blasHandle;
    hipsolverHandle_t solverHandle;
    hipsolverGesvdjInfo_t gesvdParams;
    int lwork;
    float *work;
    int batchSize = (rows / TILE_DIM) * (cols / TILE_DIM);
    int numTiles = (rows / TILE_DIM) * (cols / TILE_DIM);
    const float one = 1, zero = 0;

    CUDA_CHECK(hipMallocManaged(&info, sizeof(int) * batchSize));
    CUDA_CHECK(hipMallocManaged(&AT, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&A, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&U, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&pyU, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&V, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&pyV, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&inv, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&S, sizeof(float) * numTiles * TILE_DIM));

    int bb = myreadbin("../out/A.bin", A);

    CUDA_CHECK(hipMemPrefetchAsync(info, sizeof(int) * batchSize, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(AT, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(A, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(U, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(pyU, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(V, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(pyV, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(S, sizeof(float) * numTiles * TILE_DIM, device, stream));

    CUSOLVER_CHECK(hipsolverDnCreate(&solverHandle));
    CUSOLVER_CHECK(hipsolverSetStream(solverHandle, stream));
    CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdParams));
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdParams, 1e-5));
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdParams, 1000));
    CUBLAS_CHECK(hipblasCreate(&blasHandle));
    CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));

    // mtxtp_a100_best_param(true, rows, cols, AT, lda_T, A, lda, stream);
    // CUDA_CHECK(hipDeviceSynchronize());
    
    for(int tile_id = 0; tile_id < (cols / TILE_DIM) * (rows / TILE_DIM); ++tile_id){
        for(int i = 0; i < TILE_DIM; ++i){
            for(int j = 0; j < TILE_DIM; ++j){
                std::cout << A[i + j * TILE_DIM + tile_id * TILE_DIM * TILE_DIM] << ", ";
            }
            std::cout << "\n";
        }
        std::cout << "===================\n";
    }
    // exit(0);

    // CUDA_CHECK(hipMemcpy(A, AT, sizeof(float) * rows * cols, hipMemcpyDefault));
    // CUBLAS_CHECK(hipblasSgeam(blasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows, cols, &one, AT, lda_T, &zero, A, lda, A, lda));

    for(int i = 0; i < rows * cols; ++i){
        std::cout << A[i] << ", ";
        if((i + 1) % cols == 0){
            std::cout << "\n";
        }
    }

    CUSOLVER_CHECK(hipsolverDnSgesvdjBatched_bufferSize(solverHandle, 
                                 HIPSOLVER_EIG_MODE_VECTOR,
                                 TILE_DIM, TILE_DIM, 
                                 A, TILE_DIM, S, U, TILE_DIM, V, TILE_DIM,
                                 &lwork, gesvdParams, batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&work), sizeof(float) * lwork));

    CUSOLVER_CHECK(hipsolverDnSgesvdjBatched(solverHandle, HIPSOLVER_EIG_MODE_VECTOR, 
                TILE_DIM, TILE_DIM, 
                A, TILE_DIM, S, U, TILE_DIM, V, TILE_DIM,
                work, lwork, info, gesvdParams, batchSize));
    CUDA_CHECK(hipDeviceSynchronize());

    for(int i = 0; i < batchSize; ++i){
        if (0 == info[i]) {
            std::printf("matrix %d: gesvdj converges \n", i);
        } else if (0 > info[i]) {
            /* only info[0] shows if some input parameter is wrong.
             * If so, the error is HIPSOLVER_STATUS_INVALID_VALUE.
             */
            std::printf("Error: %d-th parameter is wrong \n", -info[i]);
            exit(1);
        } else { /* info = m+1 */
                 /* if info[i] is not zero, Jacobi method does not converge at i-th matrix. */
            std::printf("WARNING: matrix %d, info = %d : gesvdj does not converge \n", i, info[i]);
        }
    }

    std::cout << "U\n";
    print_matrix_colmaj(U, 4, 4, 4);
    std::cout << "V\n";
    print_matrix_colmaj(V, 4, 4, 4);
    std::cout << "S\n";
    print_matrix_rowmaj(S, 1, 4, 4);

    mmd_batched_a100_best_param(false, U, S, inv, batchSize);
    hipblasGemmStridedBatchedEx(
        blasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
        TILE_DIM, TILE_DIM, TILE_DIM,
        &one,
        inv, HIP_R_32F, TILE_DIM, TILE_DIM * TILE_DIM,
        V, HIP_R_32F, TILE_DIM, TILE_DIM * TILE_DIM,
        &zero,
        inv, HIP_R_32F, TILE_DIM, TILE_DIM * TILE_DIM,
        batchSize, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT
    );

    hipDeviceSynchronize();


    std::cout << "====================\nGemm from GPU\n";
    print_matrix_rowmaj(inv, 4, 4, 4);

    // print_matrix_rowmaj(pyU, 4, 4, 4);
    // print_matrix_rowmaj(pyV, 4, 4, 4);
    // print_matrix_rowmaj(S, 1, 4, 4);

    mtxtp_a100_best_param(false, rows, cols, U, lda, pyU, lda, stream);
    mtxtp_a100_best_param(false, rows, cols, V, lda, pyV, lda, stream);
    hipDeviceSynchronize();

    // CUBLAS_CHECK(hipblasSgeam(blasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, cols, rows, &one, U, ldu, &zero, pyU, ldu_T, pyU, ldu_T));
    // CUBLAS_CHECK(hipblasSgeam(blasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, cols, rows, &one, V, ldv, &zero, pyV, ldv_T, pyV, ldv_T));
    // CUDA_CHECK(hipDeviceSynchronize());

    // std::cout << "======================\nU\n";
    // print_matrix_rowmaj(V, rows, cols, lda);
    // std::cout << "======================\npyU\n";
    // print_matrix_rowmaj(pyV, rows, cols, lda);

    writebin("../out/U.bin", U, sizeof(float) * rows * cols);
    writebin("../out/V.bin", V, sizeof(float) * rows * cols);
    writebin("../out/S.bin", S, sizeof(float) * numTiles * TILE_DIM);

    // print_matrix_colmaj(A, rows, cols, lda);
    // print_matrix_colmaj(U, rows, cols, lda);
    // print_matrix_colmaj(V, rows, cols, lda);
    // print_matrix_rowmaj(S, 1, N, lda);

    std::cout << "Exit bwm with 0\n";

}

#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include <iostream>
#include "myutils.cpp"
#include "mydct.cu"
#include "constants.h"

struct myhandle {
    hipsolverHandle_t solver;
    hipblasHandle_t blas;
    int lwork;
    float *work;
};


void gesvd(size_t rows, size_t cols, float *A, size_t lda, float *S, float *U, size_t ldu, float *V, size_t ldv, hipStream_t stream=0){

    hipsolverHandle_t cusolverHandle;
    hipsolverGesvdjInfo_t gesvdinfo;
    int lwork;
    // float *work;
    int batch_size = (rows / TILE_DIM) * (cols / TILE_DIM);

    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverHandle));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverHandle, stream));
    CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdinfo));

    hipsolverDnSgesvdjBatched_bufferSize(cusolverHandle, 
                                 HIPSOLVER_EIG_MODE_VECTOR,
                                 rows, cols, 
                                 A, lda, S, U, ldu, V, ldv,
                                 &lwork, gesvdinfo, batch_size);

}


void tiled_add_wm(size_t rows, size_t cols, float *A, size_t lda, float *res, size_t ldres, float *workspace, hipStream_t stream=0){

    dct_a100_best_param(rows, cols, A, lda, workspace, ldres, stream);



    idct_a100_best_param(rows, cols, workspace, lda, res, ldres, stream);

}


__global__ void trans_and_pack_continguous(size_t rows, size_t cols, float *A, size_t lda, float *C, size_t ldc){

    // shared memory size equals to blockDim
    extern __shared__ float sA[];

    int tile_id = threadIdx.x + blockIdx.x * blockDim.x;
    int tile_per_row = cols / TILE_DIM;
    int num_tiles = (rows / TILE_DIM) * (cols / TILE_DIM);
    
    // grid stride loop
#pragma unroll
    for(; tile_id < num_tiles; tile_id += gridDim.x){

        // compute the starting address of current tile in A
        int tile_x = tile_id / tile_per_row;
        int tile_y = tile_id % tile_per_row;
        int tile_offset_to_A = tile_x * TILE_DIM * lda + tile_y * TILE_DIM;
        const float *tile_ptr_to_A = &A[tile_offset_to_A];
        
        // copy to shared memory
        sA[threadIdx.x * TILE_DIM * TILE_DIM + threadIdx.y + threadIdx.z * TILE_DIM] = 
                 tile_ptr_to_A[IDX(threadIdx.y, threadIdx.z, lda)]; // note that leading dimension is cols
        __syncthreads();

        // compute the starting address of current tile in sA
        float *tile_ptr_to_shared = &sA[threadIdx.x * TILE_DIM * TILE_DIM];
        float *elm_ptr_to_res = &C[tile_offset_to_A + threadIdx.y * ldc + threadIdx.z];

        *elm_ptr_to_res = tile_ptr_to_shared[threadIdx.y * TILE_DIM + threadIdx.z];
    }
}


int main(){
    int N = 8;
    float *dA, *dRes;
    hipMallocManaged(&dA, sizeof(float) * (N + 1) * N);
    hipMallocManaged(&dRes, sizeof(float) * (N + 1) * N);
    for (size_t i = 0; i < N; ++i) {
        for(size_t j = 0; j < N; ++j){
            dA[i + j * (N + 1)] = i + j * N;
        }
    }
    print_matrix_rowmaj(dA, N, N + 1, N + 1);
    hipMemPrefetchAsync(dA, sizeof(float) * (N + 1) * N, 0);
    hipMemPrefetchAsync(dRes, sizeof(float) * (N + 1) * N, 0);
    hipDeviceSynchronize();
    dim3 dimGrid(1024);
    dim3 dimgBlock(8, TILE_DIM, TILE_DIM);
    size_t smemSize = TILE_DIM * TILE_DIM * sizeof(int);
    for(int _iter = 0; _iter < 1; ++_iter){
        __TIMER_START__(duration)
        trans_and_pack_continguous<<<dimGrid, dimgBlock, smemSize>>>(N, N, dA, N + 1, dRes, N + 1);
        hipDeviceSynchronize();
        __TIMER_STOP__(duration);
        std::cout << "Transpose in "<< duration / 1000 << " ms\n";
    }
    print_matrix_rowmaj(dRes, N + 1, N, N + 1);
}



int maind(int argc, char **argv){

    int device = 0;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    std::cout << "Using device " << device << " " << prop.name << std::endl;

    int N = atoi(argv[1]);
    int rows = N;
    int cols = N;

    float *A, *AT, *U, *S, *V;
    int *info;
    int lda = N;
    int ldu = N;
    int ldv = N;

    hipStream_t stream = NULL;
    hipblasHandle_t blasHandle;
    hipsolverHandle_t solverHandle;
    hipsolverGesvdjInfo_t gesvdParams;
    int lwork;
    float *work;
    int batchSize = (rows / TILE_DIM) * (cols / TILE_DIM);

    CUDA_CHECK(hipMallocManaged(&info, sizeof(int) * batchSize));
    CUDA_CHECK(hipMallocManaged(&AT, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&A, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&U, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&V, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&S, sizeof(float) * batchSize * TILE_DIM));

    int bb = myreadbin("../out/A.bin", AT);

    CUDA_CHECK(hipMemPrefetchAsync(AT, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(A, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(U, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(V, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(S, sizeof(float) * N, device, stream));

    CUSOLVER_CHECK(hipsolverDnCreate(&solverHandle));
    CUSOLVER_CHECK(hipsolverSetStream(solverHandle, stream));
    CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdParams));
    CUBLAS_CHECK(hipblasCreate(&blasHandle));
    CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));

    const float one = 1, zero = 0;
    CUBLAS_CHECK(hipblasSgeam(blasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows, cols, &one, AT, lda, &zero, A, lda, A, lda));
    

    CUSOLVER_CHECK(hipsolverDnSgesvdjBatched_bufferSize(solverHandle, 
                                 HIPSOLVER_EIG_MODE_VECTOR,
                                 TILE_DIM, TILE_DIM, 
                                 A, lda, S, U, ldu, V, ldv,
                                 &lwork, gesvdParams, batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&work), sizeof(float) * lwork));

    CUSOLVER_CHECK(hipsolverDnSgesvdjBatched(solverHandle, HIPSOLVER_EIG_MODE_VECTOR, 
                TILE_DIM, TILE_DIM, 
                A, lda, S, U, ldu, V, ldv,
                work, lwork, info, gesvdParams, batchSize));
    CUDA_CHECK(hipDeviceSynchronize());

    writebin("../out/U.bin", U, sizeof(float) * rows * cols);
    writebin("../out/V.bin", V, sizeof(float) * rows * cols);
    writebin("../out/S.bin", S, sizeof(float) * batchSize * TILE_DIM);

    // print_matrix_colmaj(A, rows, cols, lda);
    // print_matrix_colmaj(U, rows, cols, lda);
    // print_matrix_colmaj(V, rows, cols, lda);
    // print_matrix_rowmaj(S, 1, N, lda);

}

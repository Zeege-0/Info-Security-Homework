#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include <iostream>
#include "myutils.cpp"
#include "mydct.cu"
#include "constants.h"

struct myhandle {
    hipsolverHandle_t solver;
    hipblasHandle_t blas;
    int lwork;
    float *work;
};


void gesvd(size_t rows, size_t cols, float *A, size_t lda, float *S, float *U, size_t ldu, float *V, size_t ldv, hipStream_t stream=0){

    hipsolverHandle_t cusolverHandle;
    hipsolverGesvdjInfo_t gesvdinfo;
    int lwork;
    // float *work;
    int batch_size = (rows / TILE_DIM) * (cols / TILE_DIM);

    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverHandle));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverHandle, stream));
    CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdinfo));

    hipsolverDnSgesvdjBatched_bufferSize(cusolverHandle, 
                                 HIPSOLVER_EIG_MODE_VECTOR,
                                 rows, cols, 
                                 A, lda, S, U, ldu, V, ldv,
                                 &lwork, gesvdinfo, batch_size);

}


void tiled_add_wm(size_t rows, size_t cols, float *A, size_t lda, float *res, size_t ldres, float *workspace, hipStream_t stream=0){

    dct_a100_best_param(rows, cols, A, lda, workspace, ldres, stream);



    idct_a100_best_param(rows, cols, workspace, lda, res, ldres, stream);

}


__global__ void gpu_trans_and_pack_continguous(size_t rows, size_t cols, float *A, size_t lda, float *C, size_t ldc){

    // shared memory size equals to blockDim
    extern __shared__ float sA[];

    int tile_id = threadIdx.x + blockIdx.x * blockDim.x;
    int tile_per_row = cols / TILE_DIM;
    int num_tiles = (rows / TILE_DIM) * (cols / TILE_DIM);
    
    // grid stride loop
#pragma unroll
    for(; tile_id < num_tiles; tile_id += gridDim.x){

        // compute the starting address of current tile in A
        int tile_x = tile_id / tile_per_row;
        int tile_y = tile_id % tile_per_row;
        const float *tile_ptr_to_A = &A[tile_x * TILE_DIM * lda + tile_y * TILE_DIM];
        float *tile_ptr_to_shared = &sA[threadIdx.x * TILE_DIM * TILE_DIM];
        float *tile_ptr_to_res = &C[tile_id * TILE_DIM * TILE_DIM];
        
        // copy to shared memory
        tile_ptr_to_shared[threadIdx.y + threadIdx.z * TILE_DIM] = 
                 tile_ptr_to_A[IDX(threadIdx.y, threadIdx.z, lda)]; // note that leading dimension is cols
        __syncthreads();

        tile_ptr_to_res[threadIdx.y * TILE_DIM + threadIdx.z] = tile_ptr_to_shared[threadIdx.y * TILE_DIM + threadIdx.z];
    }
}

__global__ void gpu_unpack_and_trans(size_t rows, size_t cols, const float *A, size_t lda, float *C, size_t ldc){
    // shared memory size equals to blockDim
    extern __shared__ float sA[];

    int tile_id = threadIdx.x + blockIdx.x * blockDim.x;
    int tile_per_row = cols / TILE_DIM;
    int num_tiles = (rows / TILE_DIM) * (cols / TILE_DIM);
    
    // grid stride loop
#pragma unroll
    for(; tile_id < num_tiles; tile_id += gridDim.x){

        // compute the starting address of current tile in A
        int tile_x = tile_id / tile_per_row;
        int tile_y = tile_id % tile_per_row;
        float *tile_ptr_to_A = &C[tile_x * TILE_DIM * lda + tile_y * TILE_DIM];
        float *tile_ptr_to_shared = &sA[threadIdx.x * TILE_DIM * TILE_DIM];
        const float *tile_ptr_to_res = &A[tile_id * TILE_DIM * TILE_DIM];
        
        tile_ptr_to_shared[threadIdx.y * TILE_DIM + threadIdx.z] = tile_ptr_to_res[threadIdx.y * TILE_DIM + threadIdx.z];
        // copy to shared memory
        tile_ptr_to_A[IDX(threadIdx.y, threadIdx.z, lda)] = tile_ptr_to_shared[threadIdx.y + threadIdx.z * TILE_DIM];
        __syncthreads();

        // printf("(%d, %d, %d): %d\n", tile_id, threadIdx.y, threadIdx.z, tile_x * TILE_DIM * lda + tile_y * TILE_DIM);

    }
}


int mtxtp_a100_best_param(bool input, size_t rows, size_t cols, float *A, size_t lda, float *C, size_t ldc, hipStream_t stream=0){
    dim3 dimGrid(1024);
    dim3 dimgBlock(8, TILE_DIM, TILE_DIM);
    size_t smemSize = TILE_DIM * TILE_DIM * sizeof(int);
    __TIMER_START__(dur);
    if (input) {
        gpu_trans_and_pack_continguous<<<dimGrid, dimgBlock, smemSize, stream>>>(rows, cols, A, lda, C, ldc);
    } else {
        gpu_unpack_and_trans<<<dimGrid, dimgBlock, smemSize, stream>>>(rows, cols, A, lda, C, ldc);
    }
    __TIMER_STOP__(dur);
    std::cout << "Trans: " << dur << std::endl;
}

int main(int argc, char **argv){

    int device = 0;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    std::cout << "Using device " << device << " " << prop.name << std::endl;

    int N = atoi(argv[1]);
    int rows = N;
    int cols = N;

    float *A, *AT, *U, *S, *V;
    float *pyU, *pyS, *pyV;
    int *info;
    int lda = N;
    int ldu = N;
    int ldv = N;

    hipStream_t stream = NULL;
    hipblasHandle_t blasHandle;
    hipsolverHandle_t solverHandle;
    hipsolverGesvdjInfo_t gesvdParams;
    int lwork;
    float *work;
    int batchSize = (rows / TILE_DIM) * (cols / TILE_DIM);

    CUDA_CHECK(hipMallocManaged(&info, sizeof(int) * batchSize));
    CUDA_CHECK(hipMallocManaged(&AT, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&A, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&U, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&pyU, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&V, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&pyV, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&S, sizeof(float) * batchSize * TILE_DIM));

    int bb = myreadbin("../out/A.bin", AT);

    CUDA_CHECK(hipMemPrefetchAsync(info, sizeof(int) * batchSize, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(AT, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(A, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(U, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(pyU, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(V, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(pyV, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(S, sizeof(float) * N, device, stream));

    CUSOLVER_CHECK(hipsolverDnCreate(&solverHandle));
    CUSOLVER_CHECK(hipsolverSetStream(solverHandle, stream));
    CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdParams));
    CUBLAS_CHECK(hipblasCreate(&blasHandle));
    CUBLAS_CHECK(hipblasSetStream(blasHandle, stream));

    mtxtp_a100_best_param(true, rows, cols, AT, lda, A, lda, stream);
    hipDeviceSynchronize();

    for(int i = 0; i < rows * cols; ++i){
        std::cout << A[i] << ", ";
        if((i + 1) % cols == 0){
            std::cout << "\n";
        }
    }
    

    CUSOLVER_CHECK(hipsolverDnSgesvdjBatched_bufferSize(solverHandle, 
                                 HIPSOLVER_EIG_MODE_VECTOR,
                                 TILE_DIM, TILE_DIM, 
                                 A, TILE_DIM, S, U, TILE_DIM, V, TILE_DIM,
                                 &lwork, gesvdParams, batchSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&work), sizeof(float) * lwork));

    CUSOLVER_CHECK(hipsolverDnSgesvdjBatched(solverHandle, HIPSOLVER_EIG_MODE_VECTOR, 
                TILE_DIM, TILE_DIM, 
                A, TILE_DIM, S, U, TILE_DIM, V, TILE_DIM,
                work, lwork, info, gesvdParams, batchSize));
    CUDA_CHECK(hipDeviceSynchronize());

    mtxtp_a100_best_param(false, rows, cols, A, lda, pyU, lda, stream);
    mtxtp_a100_best_param(false, rows, cols, V, lda, pyV, lda, stream);
    hipDeviceSynchronize();

    print_matrix_colmaj(pyU, 4, 4, 4);

    writebin("../out/U.bin", U, sizeof(float) * rows * cols);
    writebin("../out/V.bin", V, sizeof(float) * rows * cols);
    writebin("../out/S.bin", S, sizeof(float) * batchSize * TILE_DIM);

    // print_matrix_colmaj(A, rows, cols, lda);
    // print_matrix_colmaj(U, rows, cols, lda);
    // print_matrix_colmaj(V, rows, cols, lda);
    // print_matrix_rowmaj(S, 1, N, lda);

}

#include "hip/hip_runtime.h"

// #pragma once

#include "myutils.cpp"
#include "constants.h"

/**
 * Matrix multiply diagonal, the API is similar to cublasGemmBatched. 
 * Should be launched with 3D block (__, TILE_DIM, TILE_DIM) and 1D grid.
 * Matrix stored in column major.
*/
__global__ void gpu_mmd_batched(float *A, float *D, float *res, size_t batchSize){
    size_t tile_id = threadIdx.x + blockIdx.x * blockDim.x;
    for(; tile_id < batchSize; tile_id += blockDim.x){
        size_t offset = threadIdx.y + threadIdx.z * TILE_DIM + tile_id * TILE_DIM * TILE_DIM;
        res[offset] = A[offset] * D[threadIdx.y + tile_id * TILE_DIM];
    }
}

void mmd_batched_a100_best_param(float *A, float *D, float *res, size_t batchSize){
    dim3 dimGrid = dim3(512);
    dim3 dimBlock = dim3(8, TILE_DIM, TILE_DIM);
    gpu_mmd_batched<<<dimGrid, dimBlock>>>(A, D, res, batchSize);
}

int main(){

    float *A, *D, *res;
    int N = 8;
    hipMallocManaged(&A, sizeof(float) * N * N);
    hipMallocManaged(&res, sizeof(float) * N * N);
    hipMallocManaged(&D, sizeof(float) * 16);

    for(int i = 0; i < N * N; ++i){
        A[(i % N) * N + i / N] = i + 1;
    }

    D[0] = 0.001;
    for(int i = 1; i < 16; ++i){
        D[i] = D[i - 1] * 10;
    }

    mmd_batched_a100_best_param(A, D, res, 4);
    hipDeviceSynchronize();

    print_matrix_colmaj(A, N, N, N);
    print_matrix_colmaj(res, N, N, N);

}


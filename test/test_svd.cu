
#include "../src/svd.cu"


int main(int argc, char **argv){

    __TIMER_START__(end2end);

    int device = 0;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    std::cout << "Using device " << device << " " << prop.name << std::endl;

    int rows = atoi(argv[1]);
    int cols = atoi(argv[2]);
    int wmlen = atoi(argv[3]);

    float *A, *U, *S, *V, *inv;
    int mod1 = 10;
    uint8_t *wm, *wmget;
    int *info;

    hipStream_t stream = NULL;
    hipblasHandle_t blasHandle;
    hipsolverHandle_t solverHandle;
    hipsolverGesvdjInfo_t gesvdParams;
    int lwork;
    float *work;
    size_t numTiles = (rows / TILE_DIM) * (cols / TILE_DIM);

    CUDA_CHECK(hipMallocManaged(&wm, sizeof(uint8_t) * wmlen));
    CUDA_CHECK(hipMallocManaged(&wmget, sizeof(uint8_t) * wmlen));
    CUDA_CHECK(hipMallocManaged(&info, sizeof(int) * numTiles));
    CUDA_CHECK(hipMallocManaged(&A, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&U, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&V, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&inv, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&S, sizeof(float) * numTiles * TILE_DIM));

    int bb = myreadbin("../out/A.bin", A);
    bb = myreadbin("../out/wm.bin", wm);
    std::cout << "Read watermark\n";
    for(int i = 0; i < wmlen; ++i){
        std::cout << int(wm[i]) << ", ";
    }
    std::cout << "\n";

    CUDA_CHECK(hipMemPrefetchAsync(wm, sizeof(uint8_t) * wmlen, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(wmget, sizeof(uint8_t) * wmlen, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(info, sizeof(int) * numTiles, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(A, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(U, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(V, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(S, sizeof(float) * numTiles * TILE_DIM, device, stream));

    init_cudalib(&solverHandle, &blasHandle, numTiles, A, U, S, V, &work, &lwork, &gesvdParams, stream);
    std::cout << "Allocated " << lwork << " float buffer for gesvd\n";

    __TIMER_START__(computation);
    gesvd_a100_best_param(solverHandle, numTiles, A, U, S, V, work, lwork, info, gesvdParams);
    CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "Before add wm\n";
    print_matrix_rowmaj(S, 2, TILE_DIM, TILE_DIM);
    tiled_add_wm_a100_bestparam(numTiles, S, wm, wmlen, mod1, stream);
    hipDeviceSynchronize();
    std::cout << "After add wm\n";
    print_matrix_rowmaj(S, 2, TILE_DIM, TILE_DIM);
    tiled_get_wm_a100_bestparam(numTiles, S, wmget, wmlen, mod1, stream);

    mmd_batched_a100_best_param(false, U, S, inv, numTiles);
    invsvd_a100_best_param(blasHandle, numTiles, inv, U, S, V);
    hipDeviceSynchronize();
    __TIMER_STOP__(computation);


    for(int i = 0; i < numTiles; ++i){
        if (0 == info[i]) {
            // std::printf("matrix %d: gesvdj converges \n", i);
        } else if (0 > info[i]) {
            std::printf("Error: %d-th parameter is wrong \n", -info[i]);
            exit(1);
        } else {
            std::printf("WARNING: matrix %d, info = %d : gesvdj does not converge \n", i, info[i]);
        }
    }

    // std::cout << "====================\nGemm from GPU\n";
    // print_matrix_rowmaj(inv, 8, 8, 8);

    writebin("../out/inv.bin", inv, sizeof(float) * rows * cols);
    writebin("../out/wmget.bin", wmget, sizeof(uint8_t) * wmlen);

    // for(int i = 0; i < wmlen; ++i){
    //     std::cout << int(wmget[i]) << ", ";
    // }
    // std::cout << "\n";

    std::cout << "GPU computation " << computation << " ms\n";
    __TIMER_STOP__(end2end);
    std::cout << "GPU end to end " << end2end << " ms\n";

    std::cout << "Exit bwm with 0\n";

}


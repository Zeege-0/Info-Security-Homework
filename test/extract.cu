
#include <opencv2/opencv.hpp>

#include "../src/svd.cu"
#include "../src/mydct.cu"
#include "../src/dwt.cu"


int main(int argc, char **argv){

    __TIMER_START__(end2end);

    int device = 0;
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    std::cout << "Using device " << device << " " << prop.name << std::endl;

    size_t orirows = atoll(argv[1]);
    size_t oricols = atoll(argv[2]);
    size_t wmlen = atoll(argv[3]);

    size_t rows = 16 * (orirows / 32);
    size_t cols = 16 * (oricols / 32);

    printf("Ori: (%d, %d), Rnd:(%d, %d)\n", orirows, oricols, rows, cols);

    float *U, *S, *V, *inv, *dct, *wmget;
    float *Coefs, *Img;
    float *coefs[4];
    int mod1 = 37, mod2 = 11;
    uint8_t *wm;
    int *info;

    hipStream_t stream = NULL;
    hipblasHandle_t blasHandle;
    hipsolverHandle_t solverHandle;
    hipsolverGesvdjInfo_t gesvdParams;
    int lwork;
    float *work;
    size_t numTiles = (rows / TILE_DIM) * (cols / TILE_DIM);

    CUDA_CHECK(hipMallocManaged(&wm, sizeof(uint8_t) * wmlen));
    CUDA_CHECK(hipMallocManaged(&wmget, sizeof(float) * wmlen));
    CUDA_CHECK(hipMallocManaged(&info, sizeof(int) * numTiles));
    CUDA_CHECK(hipMallocManaged(&Img, sizeof(float) * orirows * oricols));
    CUDA_CHECK(hipMallocManaged(&Coefs, sizeof(float) * orirows * oricols));
    CUDA_CHECK(hipMallocManaged(&dct, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&U, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&V, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&inv, sizeof(float) * rows * cols));
    CUDA_CHECK(hipMallocManaged(&S, sizeof(float) * numTiles * TILE_DIM));
    coefs[0] = &Coefs[0 * rows * cols];
    coefs[1] = &Coefs[1 * rows * cols];
    coefs[2] = &Coefs[2 * rows * cols];
    coefs[3] = &Coefs[3 * rows * cols];
    init_cudalib(&solverHandle, &blasHandle, numTiles, coefs[0], U, S, V, &work, &lwork, &gesvdParams, stream);
    std::cout << "Finnish allocation\n";

    CHECK_READ(myreadbin("../out/gpuout.bin", Img));
    CHECK_READ(myreadbin("../out/wm.bin", wm));

    CUDA_CHECK(hipMemPrefetchAsync(wm, sizeof(uint8_t) * wmlen, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(wmget, sizeof(float) * wmlen, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(info, sizeof(int) * numTiles, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(Img, sizeof(float) * orirows * oricols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(Coefs, sizeof(float) * orirows * oricols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(dct, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(U, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(V, sizeof(float) * rows * cols, device, stream));
    CUDA_CHECK(hipMemPrefetchAsync(S, sizeof(float) * numTiles * TILE_DIM, device, stream));
    CUDA_CHECK(hipDeviceSynchronize());

    __TIMER_START__(computation);

    haar_forward2d(rows * 2, cols * 2, Img, oricols, coefs);
    CUDA_CHECK(hipDeviceSynchronize());

    dct_a100_best_param(rows, cols, coefs[0], cols, dct, cols, stream);
    CUDA_CHECK(hipDeviceSynchronize());
    writebin("../out/dct.bin", dct, sizeof(float) * rows * cols);

    gesvd_a100_best_param(solverHandle, numTiles, dct, U, S, V, work, lwork, info, gesvdParams);
    CUDA_CHECK(hipDeviceSynchronize());

    tiled_get_wm_a100_bestparam(numTiles, S, wmget, wmlen, mod1, mod2, stream);

    hipDeviceSynchronize();
    __TIMER_STOP__(computation);


    for(int i = 0; i < numTiles; ++i){
        if (0 == info[i]) {
            // std::printf("matrix %d: gesvdj converges \n", i);
        } else if (0 > info[i]) {
            std::printf("Error: %d-th parameter is wrong \n", -info[i]);
            exit(1);
        } else {
            std::printf("WARNING: matrix %d, info = %d : gesvdj does not converge \n", i, info[i]);
        }
    }

    writebin("../out/wmget.bin", wmget, sizeof(float) * wmlen);

    std::cout << "GPU computation " << computation << " ms\n";
    __TIMER_STOP__(end2end);
    std::cout << "GPU end to end " << end2end << " ms\n";

    std::cout << "Exit bwm with 0\n";

}

